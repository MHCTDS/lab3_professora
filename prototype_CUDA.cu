#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
// Nao vale a pena fazer esse calculo em gpu com so 4
// entao usei 1 milhao
// Demorou 6 mil nanosegundos para fazer a soma com 1 milhao
// O bottleneck e alocar isso tudo para a gpu, demorou 66 milhoes de nanosegundos
#define N 1000

__global__
void add(float *x,float *y,float *sum, int n) {

  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int stride=blockDim.x*gridDim.x;
  for(int i=index; i<n;i+=stride){
    *sum+=x[i]+y[i];
  }
}

int main(){

  float *x,*y,*sum;

  hipMallocManaged(&x,N*sizeof(float));
  hipMallocManaged(&y,N*sizeof(float));
  hipMallocManaged(&sum,sizeof(float));

  for(int i=0; i<N;i++){
    x[i]=1;
  }
  
  for(int i=0; i<N;i++){
    y[i]=1;
  }
  
  *sum=0;

  hipMemPrefetchAsync(x,N*sizeof(float),0,0);
  hipMemPrefetchAsync(y,N*sizeof(float),0,0);
  hipMemPrefetchAsync(sum,sizeof(float),0,0);
  
  int blockSize = 256;
  int numBlocks=(N+blockSize-1)/blockSize;
  add<<<numBlocks,blockSize>>>(x,y,sum,N);

  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  std::cout << "final sum: " << sum << std::endl;

  hipFree(x);
  hipFree(y);
  hipFree(sum);

  return 0;
}
